// Copyright 2024 Huawei Technologies Co., Ltd

// Licensed under the Apache License, Version 2.0(the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at

// http: // www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == ==
#include <stdio.h>
#include <hip/hip_runtime.h>


#define ENABLE_NOTE_LOG 0

__global__ void initArray(float *arr, const int N, const float val)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        arr[idx] = val;
    }
}

__global__ void flash_attn_1_fwd_f32_kernel(
    const float *Q,
    const float *K,
    const float *V,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float *l,
    float *m,
    float *O)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y; // batch and head index

    // Offset into Q,K,V,O,l,m - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d); // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);          // offset for l and m

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int tile_size = Bc * d; // size of Qi, Kj, Vj
    float *Qi = sram;
    float *Kj = &sram[tile_size];
    float *Vj = &sram[tile_size * 2];
    float *S = &sram[tile_size * 3];

    for (int j = 0; j < Tc; j++)
    {

        // Load Kj, Vj to SRAM
        for (int x = 0; x < d; x++)
        {
            Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];
            Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
        }
        __syncthreads(); // such that the inner loop can use the correct Kj, Vj

        for (int i = j; i < Tr; i++)
        {
            if (i * Br + tx >= N)
                break; // break if we are done with the sequence

            // Load Qi to SRAM, l and m to registers
            for (int x = 0; x < d; x++)
            {
                Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) + x];
            }
            float row_m_prev = m[lm_offset + (Br * i) + tx];
            float row_l_prev = l[lm_offset + (Br * i) + tx];

            // S = QK^T, row_m = rowmax(S)
            // S[tx][y] = Sum_{x = 0}^{d-1} {Qi[tx][x] * Kj[y][x]}
            // row_m = Max_{y = 0}^{Bc-1} S[tx][y]
            // with causal masking
            float row_m = -INFINITY;
            for (int y = 0; y < Bc; y++)
            {
                if (j * Bc + y >= N)
                    break; // break if we are done with the sequence
                float sum = 0;
                for (int x = 0; x < d; x++)
                {
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                if (i * Br + tx < j * Bc + y)
                    sum = -INFINITY;
                S[(Bc * tx) + y] = sum;

                if (sum > row_m)
                    row_m = sum;
            }

            // implement softmax with causal masking
            // P = exp(S - row_m), row_l = rowsum(P)
            // P[tx][y] = exp(S[tx][y] - row_m)
            float row_l = 0;
            for (int y = 0; y < Bc; y++)
            {
                if (j * Bc + y >= N)
                    break; // break if we are done with the sequence
                if (i * Br + tx < j * Bc + y)
                    S[(Bc * tx) + y] = 0;
                else
                    S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m);
                row_l += S[(Bc * tx) + y];
            }

            // Compute new m and l
            float row_m_new = max(row_m_prev, row_m);
            float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + (__expf(row_m - row_m_new) * row_l);

            // Write O, l, m to HBM
            for (int x = 0; x < d; x++)
            {
                float pv = 0; // Pij * Vj
                for (int y = 0; y < Bc; y++)
                {
                    if (j * Bc + y >= N)
                        break; // break if we are done with the sequence
                    pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (tx * d) + x] = (1 / row_l_new) * ((row_l_prev * __expf(row_m_prev - row_m_new) * O[qkv_offset + (tile_size * i) + (tx * d) + x]) + (__expf(row_m - row_m_new) * pv));
                // assert(!isnan(O[qkv_offset + (tile_size * i) + (tx * d) + x]));
            }
            m[lm_offset + (Br * i) + tx] = row_m_new;
            l[lm_offset + (Br * i) + tx] = row_l_new;
        }
        __syncthreads(); // otherwise, thread can use the wrong Kj, Vj in inner loop
    }
}

__global__ void flash_attn_2_fwd_f32_kernel(
    const float *Q,
    const float *K,
    const float *V,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float *L,
    float *O)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y; // batch and head index

    // Offset into Q,K,V,O - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d); // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);          // offset for L

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int tile_size = Bc * d; // size of Qi, Kj, Vj
    float *Qi = sram;
    float *Kj = &sram[tile_size];
    float *Vj = &sram[tile_size * 2];
    float *S = &sram[tile_size * 3];

    for (int i = 0; i < Tr; ++i)
    {
        if (i * Br + tx >= N)
            break; // break if we are done with the sequence

        // Load Qi from HBM to SRAM, l and m to registers
        for (int x = 0; x < d; x++)
        {
            Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) + x];
        }
        float row_m_prev = -INFINITY;
        float row_l_prev = 0;

        // Causal mask: j <= i
        for (int j = 0; j <= Tc; ++j)
        {
            __syncthreads();
            // Load Kj, Vj from HBM to SRAM
            for (int x = 0; x < d; x++)
            {
                Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];
                Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
            }
            __syncthreads();
            // S_i^j = softmax_scale * QiKj^T
            // S_i^j[tx][y] = softmax_scale * Sum_{x = 0}^{d-1} Qi[tx][x] * Kj[y][x]
            float row_m = -INFINITY;
            for (int y = 0; y < Bc; y++)
            {
                if (j * Bc + y >= N)
                    break; // break if we are done with the sequence
                if (i * Br + tx < j * Bc + y)
                    break;
                float sum = 0;
                for (int x = 0; x < d; x++)
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                sum *= softmax_scale;
                S[(Bc * tx) + y] = sum;

                if (sum > row_m)
                    row_m = sum;
            }

            // m_i^j = max(m_i^j-1, row_max(S_i^j))
            float new_row_m = max(row_m_prev, row_m);

            // P_i^j = exp(S_i^j - m_i^j)
            // P_i^j[tx][y] = exp(S_i^j[tx][y] - m_i^j)
            float row_l = 0;
            for (int y = 0; y < Bc; y++)
            {
                if (j * Bc + y >= N)
                    break; // break if we are done with the sequence
                if (i * Br + tx < j * Bc + y)
                    break;
                S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - new_row_m);
                row_l += S[(Bc * tx) + y];
            }

            // l_i^j = (exp(m_i^j-1 - m_i^j) * l_i^j-1) + row_sum(P_i^j)
            float row_m_exp = __expf(row_m_prev - new_row_m);
            float new_row_l = (row_m_exp * row_l_prev) + row_l;

            // O_i^j = diag(exp(m_i^j-1 - m_i^j))^-1 * O_i^j-1 + P_i^jVj
            for (int x = 0; x < d; x++)
            {
                float pv = 0; // Pij * Vj
                for (int y = 0; y < Bc; y++)
                {
                    if (j * Bc + y >= N)
                        break; // break if we are done with the sequence
                    if (i * Br + tx < j * Bc + y)
                        break;
                    pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (tx * d) + x] =
                    row_m_exp * O[qkv_offset + (tile_size * i) + (tx * d) + x] + pv;
            }

            // Update m and l
            row_m_prev = new_row_m;
            row_l_prev = new_row_l;
        }

        // O_i = diag(l_i^{Tc})^-1 * O_i^{Tc}
        for (int x = 0; x < d; x++)
            O[qkv_offset + (tile_size * i) + (tx * d) + x] /= row_l_prev;
        // L_i = m_i^{Tc} + log(l_i^{Tc})
        L[lm_offset + (Br * i) + tx] = row_m_prev + __logf(row_l_prev);
    }
}

__global__ void flash_attn_1_bwd_f32_kernel(
    const float *Q,
    const float *K,
    const float *V,
    const float *O,
    const float *dO,
    const float *l,
    const float *m,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float *dQ,
    float *dK,
    float *dV)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y; // batch and head index

    // Offset into Q,K,V,O,l,m - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d); // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);          // offset for l and m

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int col_tile_size = Bc * d; // size of Kj, Vj
    int row_tile_size = Br * d; // size of Qi
    float *Kj = sram;
    float *Vj = &sram[col_tile_size];

    float *dKj = &sram[col_tile_size * 2];
    float *dVj = &sram[col_tile_size * 3];

    float *Qi = &sram[col_tile_size * 4];
    float *Oi = &sram[col_tile_size * 4 + row_tile_size];
    float *dOi = &sram[col_tile_size * 4 + row_tile_size * 2];

    // We also use S for P. Likewise, we use dS for dP.
    // We can reuse the same memory because we don't need S and P at the same time.
    // We also don't need dS and dP at the same time.
    float *S = &sram[col_tile_size * 4 + row_tile_size * 3];
    float *dS = &sram[col_tile_size * 4 + row_tile_size * 3 + Bc * Br];

    for (int j = 0; j < Tc; j++)
    {

        // Load Kj, Vj to SRAM
        for (int x = 0; x < d; x++)
        {
            Kj[(tx * d) + x] = K[qkv_offset + (col_tile_size * j) + (tx * d) + x];
            Vj[(tx * d) + x] = V[qkv_offset + (col_tile_size * j) + (tx * d) + x];
        }

        // Initialize dKj, dVj to 0
        for (int x = 0; x < d; x++)
        {
            dKj[(tx * d) + x] = 0;
            dVj[(tx * d) + x] = 0;
        }

        for (int i = j; i < Tr; i++)
        {
            __syncthreads();
            // Load Qi, Oi, dOi, dQi, li, mi to SRAM
            // Also load l, m to registers
            for (int x = 0; x < d; x++)
            {
                Qi[(tx * d) + x] = Q[qkv_offset + (row_tile_size * i) + (tx * d) + x];
                Oi[(tx * d) + x] = O[qkv_offset + (row_tile_size * i) + (tx * d) + x];
                dOi[(tx * d) + x] = dO[qkv_offset + (row_tile_size * i) + (tx * d) + x];
            }
            float m_curr = m[lm_offset + (Br * i) + tx];
            float l_curr = l[lm_offset + (Br * i) + tx];

            // Sij = softmax_scale * QiKj^T
            // Sij[tx][y] = softmax_scale * Sum_{y = 0}^{Bc-1} Qi[tx][x] * Kj[y][x]
            for (int y = 0; y < Bc; y++)
            {
                float sum = 0;
                for (int x = 0; x < d; x++)
                {
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                if (i * Br + tx < j * Bc + y)
                    sum = -INFINITY;
                S[(Bc * tx) + y] = sum;
            }

            // Pij = diag(li)^-1 * exp(Sij - mi)
            // Pij[tx][y] = (1 / li[tx]) * exp(Sij[tx][y] - mi[tx])
            for (int y = 0; y < Bc; y++)
            {
                if (i * Br + tx < j * Bc + y)
                    S[(Bc * tx) + y] = 0;
                else
                    S[(Bc * tx) + y] = (1 / l_curr) * __expf(S[(Bc * tx) + y] - m_curr);
            }
            __syncthreads();
            // dVj <- dVj + Pij^T * dOi
            // dVj[tx][x] = dVj[tx][x] + Sum_{y = 0}^{Br-1} Pij[y][tx] * dOi[tx][x]
            for (int x = 0; x < d; x++)
            {
                float sum = 0;
                for (int y = 0; y < Br; y++)
                {
                    sum += S[(Bc * y) + tx] * dOi[(tx * d) + x];
                }
                atomicAdd(&dVj[(tx * d) + x], sum);
            }

            // dPij <- dOi * Vj^T
            // dPij[tx][y] = Sum_{x = 0}^{d-1} dOi[tx][x] * Vj[y][x]
            for (int y = 0; y < Bc; y++)
            {
                float sum = 0;
                for (int x = 0; x < d; x++)
                {
                    sum += dOi[(tx * d) + x] * Vj[(y * d) + x];
                }
                dS[(Bc * tx) + y] = sum;
            }

            // Di <- rowsum(dOi * Oi)  (pointwise multiply)
            // Di[tx] = Sum_{x = 0}^{d-1} dOi[tx][x] * Oi[tx][x]
            float Di = 0;
            for (int x = 0; x < d; x++)
            {
                Di += dOi[(tx * d) + x] * Oi[(tx * d) + x];
            }

            // dSij <- Pij * (dPij - Di)
            // dSij[tx][y] = Pij[tx][y] * (dPij[tx][y] - Di[tx])
            for (int y = 0; y < Bc; ++y)
            {
                dS[(Bc * tx) + y] = S[(Bc * tx) + y] * (dS[(Bc * tx) + y] - Di);
            }

            // dQi <- dQi + softmax_scale * dSijKj
            // dQ[tx][x] = dQ[tx][x] + softmax_scale * Sum_{y = 0}^{Bc-1} dSij[tx][y] * Kj[y][x]
            for (int x = 0; x < d; x++)
            {
                float sum = 0;
                for (int y = 0; y < Bc; y++)
                {
                    sum += dS[(Bc * tx) + y] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                atomicAdd(&dQ[qkv_offset + (row_tile_size * i) + (tx * d) + x], sum);
            }
            __syncthreads();
            // dKj <- dKj + softmax_scale * dSij^TQi
            // dKj[tx][x] = dKj[tx][x] + softmax_scale * Sum_{y = 0}^{Br-1} dSij[y][tx] * Qi[y][x]
            for (int x = 0; x < d; x++)
            {
                float sum = 0;
                for (int y = 0; y < Br; y++)
                {
                    sum += dS[(Bc * y) + tx] * Qi[(y * d) + x];
                }
                sum *= softmax_scale;
                atomicAdd(&dKj[(tx * d) + x], sum);
            }
        }

        // Upload Kj, Vj to HRAM
        for (int x = 0; x < d; x++)
        {
            dK[qkv_offset + (row_tile_size * j) + (tx * d) + x] = dKj[(tx * d) + x];
            dV[qkv_offset + (row_tile_size * j) + (tx * d) + x] = dVj[(tx * d) + x];
        }
    }
}

__global__ void flash_attn_2_bwd_f32_kernel(
    const float *Q,
    const float *K,
    const float *V,
    const float *O,
    const float *dO,
    const float *L,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float *dQ,
    float *dK,
    float *dV)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y; // batch and head index

    // Offset into Q,K,V,O - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d); // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);          // offset for L

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int col_tile_size = Bc * d; // size of Kj, Vj
    int row_tile_size = Br * d; // size of Qi
    float *Kj = sram;
    float *Vj = &sram[col_tile_size];

    float *dKj = &sram[col_tile_size * 2];
    float *dVj = &sram[col_tile_size * 3];

    float *Qi = &sram[col_tile_size * 4];
    float *Oi = &sram[col_tile_size * 4 + row_tile_size];
    float *dOi = &sram[col_tile_size * 4 + row_tile_size * 2];

    // We also use S for P. Likewise, we use dS for dP.
    // We can reuse the same memory because we don't need S and P at the same time.
    // We also don't need dS and dP at the same time.
    float *S = &sram[col_tile_size * 4 + row_tile_size * 3];
    float *dS = &sram[col_tile_size * 4 + row_tile_size * 3 + Bc * Br];

    for (int j = 0; j < Tc; j++)
    {

        // Load Kj, Vj to SRAM
        for (int x = 0; x < d; x++)
        {
            Kj[(tx * d) + x] = K[qkv_offset + (col_tile_size * j) + (tx * d) + x];
            Vj[(tx * d) + x] = V[qkv_offset + (col_tile_size * j) + (tx * d) + x];
        }

        // Initialize dKj, dVj to 0
        for (int x = 0; x < d; x++)
        {
            dKj[(tx * d) + x] = 0;
            dVj[(tx * d) + x] = 0;
        }

        for (int i = j; i < Tr; i++)
        {
            __syncthreads();
            // Load Qi, Oi, dOi, dQi, li, mi to SRAM
            // Also load l, m to registers
            float Di = 0;
            for (int x = 0; x < d; x++)
            {
                Qi[(tx * d) + x] = Q[qkv_offset + (row_tile_size * i) + (tx * d) + x];
                Oi[(tx * d) + x] = O[qkv_offset + (row_tile_size * i) + (tx * d) + x];
                dOi[(tx * d) + x] = dO[qkv_offset + (row_tile_size * i) + (tx * d) + x];
                Di += dOi[(tx * d) + x] * Oi[(tx * d) + x];
            }
            float l_curr = L[lm_offset + (Br * i) + tx];

            // Sij = softmax_scale * QiKj^T
            // Sij[tx][y] = softmax_scale * Sum_{y = 0}^{Bc-1} Qi[tx][x] * Kj[y][x]
            for (int y = 0; y < Bc; y++)
            {
                float sum = 0;
                for (int x = 0; x < d; x++)
                {
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                if (i * Br + tx < j * Bc + y)
                    sum = -INFINITY;
                S[(Bc * tx) + y] = sum;
            }

            // Pij = diag(li)^-1 * exp(Sij - mi)
            // Pij[tx][y] = (1 / li[tx]) * exp(Sij[tx][y] - mi[tx])
            for (int y = 0; y < Bc; y++)
            {
                if (i * Br + tx < j * Bc + y)
                    S[(Bc * tx) + y] = 0;
                else
                    S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - l_curr);
            }
            __syncthreads();
            // dVj <- dVj + Pij^T * dOi
            // dVj[tx][x] = dVj[tx][x] + Sum_{y = 0}^{Br-1} Pij[y][tx] * dOi[tx][x]
            for (int x = 0; x < d; x++)
            {
                float sum = 0;
                for (int y = 0; y < Br; y++)
                {
                    sum += S[(Bc * y) + tx] * dOi[(tx * d) + x];
                }
                atomicAdd(&dVj[(tx * d) + x], sum);
            }

            // dPij <- dOi * Vj^T
            // dPij[tx][y] = Sum_{x = 0}^{d-1} dOi[tx][x] * Vj[y][x]
            for (int y = 0; y < Bc; y++)
            {
                float sum = 0;
                for (int x = 0; x < d; x++)
                {
                    sum += dOi[(tx * d) + x] * Vj[(y * d) + x];
                }
                dS[(Bc * tx) + y] = sum;
            }

            // dSij <- Pij * (dPij - Di)
            // dSij[tx][y] = Pij[tx][y] * (dPij[tx][y] - Di[tx])
            for (int y = 0; y < Bc; ++y)
            {
                dS[(Bc * tx) + y] = S[(Bc * tx) + y] * (dS[(Bc * tx) + y] - Di);
            }

            // dQi <- dQi + softmax_scale * dSijKj
            // dQ[tx][x] = dQ[tx][x] + softmax_scale * Sum_{y = 0}^{Bc-1} dSij[tx][y] * Kj[y][x]
            for (int x = 0; x < d; x++)
            {
                float sum = 0;
                for (int y = 0; y < Bc; y++)
                {
                    sum += dS[(Bc * tx) + y] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                atomicAdd(&dQ[qkv_offset + (row_tile_size * i) + (tx * d) + x], sum);
            }
            __syncthreads();
            // dKj <- dKj + softmax_scale * dSij^TQi
            // dKj[tx][x] = dKj[tx][x] + softmax_scale * Sum_{y = 0}^{Br-1} dSij[y][tx] * Qi[y][x]
            for (int x = 0; x < d; x++)
            {
                float sum = 0;
                for (int y = 0; y < Br; y++)
                {
                    sum += dS[(Bc * y) + tx] * Qi[(y * d) + x];
                }
                sum *= softmax_scale;
                atomicAdd(&dKj[(tx * d) + x], sum);
            }
        }

        // Upload Kj, Vj to HRAM
        for (int x = 0; x < d; x++)
        {
            dK[qkv_offset + (row_tile_size * j) + (tx * d) + x] = dKj[(tx * d) + x];
            dV[qkv_offset + (row_tile_size * j) + (tx * d) + x] = dVj[(tx * d) + x];
        }
    }
}

extern "C"
{

    int flash_attn_1_fwd_f32(int nparam, void **params, int *ndims, int64_t **shapes, const char **dtypes, void *stream,
                             void *extra)
    {
        hipStream_t custream = static_cast<hipStream_t>(stream);
        if (nparam != 6)
            return 1;
        float *Q = static_cast<float *>(params[0]);
        float *K = static_cast<float *>(params[1]);
        float *V = static_cast<float *>(params[2]);
        float *O = static_cast<float *>(params[3]);
        float *l = static_cast<float *>(params[4]);
        float *m = static_cast<float *>(params[5]);

        const int B = static_cast<int>(shapes[0][0]);
        const int nh = static_cast<int>(shapes[0][1]);
        const int N = static_cast<int>(shapes[0][2]);
        const int d = static_cast<int>(shapes[0][3]);

        // initialize l to 0 and m to -inf
        hipMemset(l, 0, B * nh * N * sizeof(float));
        hipMemset(O, 0, B * nh * N * d * sizeof(float));
        int blockSize = 32;
        int numBlocks = (B * nh * N + blockSize - 1) / blockSize;
        initArray<<<numBlocks, blockSize>>>(m, B * nh * N, -INFINITY);

        // set block size, TODO: dynamically set block size
        const int Bc = 32;
        const int Br = 32;
        // const int Bc = ceil(max_sram_size / (4 * d * sizeof(float)));
        // const int Br = min(Bc, d);

        const int Tc = ceil((float)N / Bc);
        const int Tr = ceil((float)N / Br);
        const float softmax_scale = 1.0 / sqrt(d);

        // Calculate SRAM size needed per block
        int col_tile_size = Bc * d; // size of Kj, Vj
        int row_tile_size = Br * d; // size of Qi
        const int sram_size =
            (2 * col_tile_size * sizeof(float)) // SRAM size for Kj, Vj
            + (row_tile_size * sizeof(float))   // SRAM size for Qi
            + (Bc * Br * sizeof(float));        // SRAM size for S
        int max_sram_size;
        hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
        printf("Bc: %d, Br: %d, Tc: %d, Tr: %d \n", Bc, Br, Tc, Tr);
        printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

        dim3 grid_dim(B, nh); // batch_size x num_heads
        dim3 block_dim(Bc);   // Bc threads per block

        flash_attn_1_fwd_f32_kernel<<<grid_dim, block_dim, sram_size, custream>>>(
            Q, K, V, N, d, Tc, Tr, Bc, Br, softmax_scale, l, m, O);
        return 0;
    }
}

extern "C"
{

    int flash_attn_2_fwd_f32(int nparam, void **params, int *ndims, int64_t **shapes, const char **dtypes, void *stream,
                             void *extra)
    {
        hipStream_t custream = static_cast<hipStream_t>(stream);
        if (nparam != 5)
            return 1;
        float *Q = static_cast<float *>(params[0]);
        float *K = static_cast<float *>(params[1]);
        float *V = static_cast<float *>(params[2]);
        float *O = static_cast<float *>(params[3]);
        float *L = static_cast<float *>(params[4]);

        const int B = static_cast<int>(shapes[0][0]);
        const int nh = static_cast<int>(shapes[0][1]);
        const int N = static_cast<int>(shapes[0][2]);
        const int d = static_cast<int>(shapes[0][3]);

        // initialize l to 0 and m to -inf
        hipMemset(L, 0, B * nh * N * sizeof(float));
        hipMemset(O, 0, B * nh * N * d * sizeof(float));

        // set block size, TODO: dynamically set block size
        const int Bc = 32;
        const int Br = 32;

        // Calculate SRAM size needed per block
        int col_tile_size = Bc * d; // size of Kj, Vj
        int row_tile_size = Br * d; // size of Qi
        const int sram_size =
            (2 * col_tile_size * sizeof(float)) // SRAM size for Kj, Vj
            + (row_tile_size * sizeof(float))   // SRAM size for Qi
            + (Bc * Br * sizeof(float));        // SRAM size for S
        int max_sram_size;
        hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

        const int Tc = ceil((float)N / Bc);
        const int Tr = ceil((float)N / Br);
        const float softmax_scale = 1.0 / sqrt(d);

        printf("Bc: %d, Br: %d, Tc: %d, Tr: %d \n", Bc, Br, Tc, Tr);
        printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

        dim3 grid_dim(B, nh); // batch_size x num_heads
        dim3 block_dim(Bc);   // Bc threads per block

        flash_attn_2_fwd_f32_kernel<<<grid_dim, block_dim, sram_size, custream>>>(
            Q, K, V, N, d, Tc, Tr, Bc, Br, softmax_scale, L, O);
        return 0;
    }
}

extern "C"
{
    int flash_attn_1_bwd_f32(int nparam, void **params, int *ndims, int64_t **shapes, const char **dtypes, void *stream,
                             void *extra)
    {
        hipStream_t custream = static_cast<hipStream_t>(stream);
        if (nparam != 10)
            return 1;
        float *Q = static_cast<float *>(params[0]);
        float *K = static_cast<float *>(params[1]);
        float *V = static_cast<float *>(params[2]);
        float *O = static_cast<float *>(params[3]);
        float *dO = static_cast<float *>(params[4]);
        float *l = static_cast<float *>(params[5]);
        float *m = static_cast<float *>(params[6]);
        float *dQ = static_cast<float *>(params[7]);
        float *dK = static_cast<float *>(params[8]);
        float *dV = static_cast<float *>(params[9]);

        const int B = static_cast<int>(shapes[0][0]);
        const int nh = static_cast<int>(shapes[0][1]);
        const int N = static_cast<int>(shapes[0][2]);
        const int d = static_cast<int>(shapes[0][3]);

        hipMemset(dQ, 0, B * nh * N * d * sizeof(float));
        hipMemset(dK, 0, B * nh * N * d * sizeof(float));
        hipMemset(dV, 0, B * nh * N * d * sizeof(float));

        // set block size, TODO: dynamically set block size
        const int Bc = 16;
        const int Br = 16;

        // Calculate SRAM size needed per block
        int col_tile_size = Bc * d; // size of Kj, Vj
        int row_tile_size = Br * d; // size of Qi
        const int sram_size =
            (4 * col_tile_size * sizeof(float))   // SRAM size for Kj, Vj
            + (3 * row_tile_size * sizeof(float)) // SRAM size for Qi
            + (2 * Bc * Br * sizeof(float));      // SRAM size for S
        int max_sram_size;
        hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

        const int Tc = ceil((float)N / Bc);
        const int Tr = ceil((float)N / Br);
        const float softmax_scale = 1.0 / sqrt(d);

        printf("Bc: %d, Br: %d, Tc: %d, Tr: %d \n", Bc, Br, Tc, Tr);
        printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

        dim3 grid_dim(B, nh); // batch_size x num_heads
        dim3 block_dim(Bc);   // Bc threads per block

        flash_attn_1_bwd_f32_kernel<<<grid_dim, block_dim, sram_size, custream>>>(
            Q, K, V, O, dO, l, m, N, d, Tc, Tr, Bc, Br, softmax_scale, dQ, dK, dV);
        return 0;
    }
}

extern "C"
{
    int flash_attn_2_bwd_f32(int nparam, void **params, int *ndims, int64_t **shapes, const char **dtypes, void *stream,
                             void *extra)
    {
        hipStream_t custream = static_cast<hipStream_t>(stream);
        if (nparam != 9)
            return 1;
        float *Q = static_cast<float *>(params[0]);
        float *K = static_cast<float *>(params[1]);
        float *V = static_cast<float *>(params[2]);
        float *O = static_cast<float *>(params[3]);
        float *dO = static_cast<float *>(params[4]);
        float *L = static_cast<float *>(params[5]);
        float *dQ = static_cast<float *>(params[6]);
        float *dK = static_cast<float *>(params[7]);
        float *dV = static_cast<float *>(params[8]);

        const int B = static_cast<int>(shapes[0][0]);
        const int nh = static_cast<int>(shapes[0][1]);
        const int N = static_cast<int>(shapes[0][2]);
        const int d = static_cast<int>(shapes[0][3]);

        hipMemset(dQ, 0, B * nh * N * d * sizeof(float));
        hipMemset(dK, 0, B * nh * N * d * sizeof(float));
        hipMemset(dV, 0, B * nh * N * d * sizeof(float));

        // set block size, TODO: dynamically set block size
        const int Bc = 16;
        const int Br = 16;

        // Calculate SRAM size needed per block
        int col_tile_size = Bc * d; // size of Kj, Vj
        int row_tile_size = Br * d; // size of Qi
        const int sram_size =
            (4 * col_tile_size * sizeof(float))   // SRAM size for Kj, Vj
            + (3 * row_tile_size * sizeof(float)) // SRAM size for Qi
            + (2 * Bc * Br * sizeof(float));      // SRAM size for S
        int max_sram_size;
        hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

        const int Tc = ceil((float)N / Bc);
        const int Tr = ceil((float)N / Br);
        const float softmax_scale = 1.0 / sqrt(d);

        printf("Bc: %d, Br: %d, Tc: %d, Tr: %d \n", Bc, Br, Tc, Tr);
        printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

        dim3 grid_dim(B, nh); // batch_size x num_heads
        dim3 block_dim(Bc);   // Bc threads per block

        flash_attn_2_bwd_f32_kernel<<<grid_dim, block_dim, sram_size, custream>>>(
            Q, K, V, O, dO, L, N, d, Tc, Tr, Bc, Br, softmax_scale, dQ, dK, dV);
        return 0;
    }
}